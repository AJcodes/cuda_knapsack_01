#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <iostream>

#define n 10
#define W 100

hipError_t knapsackCuda(int *c, const int *a, const int *b, unsigned int size);

__device__ int maxi(int a, int b) { 
	return (a > b)? a : b; 
}

__global__ void knapsackKernel(int *wt, int *val, int *output, int i) {
	int w = threadIdx.x;

	__syncthreads();
	if (i == 0 || w == 0)
		output[(i*W)+w] = 0;
	else if (wt[i-1] <= w)
		output[(i*W)+w] = maxi(val[i-1] + output[((i-1)*W)+(w-wt[i-1])],  output[((i-1)*W)+w]);
	else
		output[(i*W)+w] = output[((i-1)*W)+w];
	__syncthreads();
   
}

int main() {
    const int val[] = { 60, 100, 120, 80, 90, 110, 70, 50, 130, 40 };
    const int wt[] = { 10, 20, 30, 40, 10, 20, 30, 40, 10, 30 };
	int *output = 0;

	output = (int *)malloc((n+1)*(W+1)*sizeof(int));

    hipError_t cudaStatus = knapsackCuda(output, val, wt, n);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "knapsackCuda failed!");
        return 1;
    }

	/*for (int i = 0; i <= n; i++)
		for (int j = 0; j <= W; j++) {
			std::cout << output[(i*W) + j] << " ";
			if (j == W)
				std::cout << std::endl;
	}*/

	std::cout << "Maxmimum Value possible for knapsack with capacity " << W << " is : " << output[(n+1)*W] << std::endl;

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t knapsackCuda(int *output, const int *val, const int *wt, unsigned int size) {
    int *dev_val = 0;
    int *dev_wt = 0;
    int *dev_output = 0;
	int i = 0;
    hipError_t cudaStatus;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_output, (size + 1) * (W + 1) * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc 1 failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_val, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc 2 failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_wt, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc 3 failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_val, val, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 1 failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_wt, wt, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 2 failed!");
        goto Error;
    }

	hipEventRecord(start);
	while (i <= n) {
		knapsackKernel<<<1, W + 1>>>(dev_wt, dev_val, dev_output, i);
		i++;
	}
	hipEventRecord(stop);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "knapsackKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching knapsackKernel!\n", cudaStatus);
        goto Error;
    }

    cudaStatus = hipMemcpy(output, dev_output, (size + 1) * (W + 1) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy 4 failed!");
        goto Error;
    }

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	std::cout << "Execution Time : " << milliseconds / 1000 << " seconds" << std::endl;

Error:
    hipFree(dev_output);
    hipFree(dev_val);
    hipFree(dev_wt);
    
    return cudaStatus;
}
